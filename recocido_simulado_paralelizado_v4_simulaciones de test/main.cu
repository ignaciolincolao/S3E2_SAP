#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <ctime>
#include <random>
#include <cstdlib>
#include <cstdint>
#include <fstream>
#include <string>
#include <sstream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include "UTM.h"
#include <iomanip>

///////////////////////////////////////////////////
/// Variables constantes CUDA
///////////////////////////////////////////////////

__constant__ int d_cupoArray[85];
__constant__ double d_alpha[3];



///////////////////////////////////////////////////
/// Estructura de datos de los colegios.
///////////////////////////////////////////////////
struct Info_colegio {
    double latitude = 0;
    double longitude = 0;
    int num_alu = 0;
    int rbd = 0;
    int prioritario = 0;
};

///////////////////////////////////////////////////
/// Estructura de alumnos
///////////////////////////////////////////////////

struct Info_alu{
    int rbd = 0;
    int sep = 0;
    double latitude = 0.0;
    double longitude = 0.0;
};


///////////////////////////////////////////////////
/// Funciones generales
///////////////////////////////////////////////////

double calCosto(const int currentThreadSolution[]);
double meanDist(const int currentThreadSolution[]);
double S(const int currentThreadSolution[]);
double costCupo(const int currentThreadSolution[]);
int acepta(double costPrevious, double costCurrent);
double p(double costPrevious,double costCurrent);
void assignSchoolToArray(Info_colegio *ptr_colegios, Info_alu *ptr_students);
void calcDist(Info_colegio *ptr_colegios, Info_alu *ptr_students);
void shuffle(int[],int,std::uniform_int_distribution<int>);

///////////////////////////////////////////////////
/// Kernel newSolution_kerneln_colegios
///////////////////////////////////////////////////


__global__ void newSolution_kernel(
        double *d_array_current_Solution,
        int *d_array_current_Solution_thread,
        const int n_students,
        const int n_colegios,
        const int n_thread,
        const double max_dist,
        const int* __restrict__ d_alumnosSep,
        const int totalVuln,
        const int* __restrict__ d_aluxcol,
        const int* __restrict__ d_aluVulxCol,
        const int* __restrict__ d_currentSolution,
        const double* __restrict__ d_distMat,
        const int* __restrict__ d_shuffle_students,
        const int* __restrict__ d_shuffle_colegios,
        size_t pitch){

    /// Shared Memory
    extern __shared__ double sharedMem[];
    int* aluxcolblock = (int*)sharedMem;
    int* aluVulxColblock = (int*)&aluxcolblock[n_colegios];
    double* solutions =(double*)&aluVulxColblock[n_colegios];
    int* solutions_thread = (int*)&solutions[n_thread];
    /// Inicializa variables en 0
    int aluchange,
            colchange,
            i = 0,
            x = 0,
            aluVulCol= 0,
            aluNoVulCol= 0,
            totalAluCol= 0,
            myID = threadIdx.x,
            school_alu_change,
            salto= n_thread;

    double  totalcostCupo= 0.0,
            totalSesc= 0.0,
            var1,
            var2,
            var3,
            result= 0.0;
    /// Inicializa arrays
    aluchange = d_shuffle_students[blockIdx.x];
    colchange = d_shuffle_colegios[threadIdx.x];
    solutions_thread[threadIdx.x] = colchange;

    /// Recopila la informacion que existe en memoria global
    clock_t start_time = clock();
    /// a shared memory29853
    school_alu_change = d_currentSolution[aluchange];
    for (i = threadIdx.x; i< n_colegios; i=i+n_thread){
        aluxcolblock[i] = d_aluxcol[i];
        aluVulxColblock[i] = d_aluVulxCol[i];
        if(i == school_alu_change){
            aluxcolblock[school_alu_change]-=1;
            aluVulxColblock[school_alu_change]-=d_alumnosSep[aluchange];
        }
    }

    /// Calcula la distancia total
    for (x = 0 ; x < n_students ; x++) {
        if (x != aluchange) {
            result += d_distMat[x * pitch / sizeof(double) + d_currentSolution[x]];
        }
        else {
            result += d_distMat[x * pitch / sizeof(double) + colchange];
        }
    }
    __syncthreads();
    clock_t stop_time = clock();
    int valtime = (int)(stop_time - start_time);
    if(threadIdx.x==0 && blockIdx.x==0){
        printf("%d \n",valtime);
    }
    /// Calcula el costo cupo y la cantidad de segregación total
    for(int n=0; n<n_colegios; n++){
        totalAluCol = aluxcolblock[n];
        aluVulCol = aluVulxColblock[n];
        if(n == colchange){
            totalAluCol+=1;
            aluVulCol+=d_alumnosSep[aluchange];
        }
        aluNoVulCol =totalAluCol - aluVulCol;
        // Calcula el costo cupo
        totalcostCupo+=totalAluCol*fabs((d_cupoArray[n]-totalAluCol)/pow(((double)d_cupoArray[n]/2),2));
        // Calcula el total sesc
        totalSesc+=((double)1/2)*fabs((aluVulCol/(double)totalVuln)-(aluNoVulCol/(double)(n_students-totalVuln)));
    }

    var1 = d_alpha[0]*((result/(double(n_students)))/double(max_dist));
    var2 = d_alpha[1]*totalSesc;
    var3 = d_alpha[2]*(totalcostCupo/n_colegios);
    solutions[myID] = var1+var2+var3;
    if(colchange == school_alu_change){
        solutions[myID] = 1.0;
    }
    __syncthreads();
    while(salto){
        if(salto-(myID+1)>myID){
            if(solutions[myID]>solutions[salto-(myID+1)]){
                solutions[myID]=solutions[salto-(myID+1)];
                solutions_thread[myID]=solutions_thread[salto-(myID+1)];
            }
        }
        salto = (salto/2)+(salto&(2-1));
        if(salto==1){
            salto = 0;
        }
        __syncthreads();
    }
    if(myID==0)
    {
        d_array_current_Solution[blockIdx.x] = solutions[myID];
        d_array_current_Solution_thread[blockIdx.x] = solutions_thread[myID];

    }
}

__global__ void reduce_block_kernel(
        double *d_array_current_Solution,
        int *d_array_current_Solution_thread,
        int *d_array_current_Solution_block,
        const int n_block){

    extern __shared__ double sharedMem[];
    double* solutions =(double*)sharedMem;
    int* solutions_block = (int*)&solutions[n_block];
    int* solutions_thread = (int*)&solutions_block[n_block];

    int myID = threadIdx.x;
    int salto= n_block;
    solutions[myID] = d_array_current_Solution[myID];
    solutions_thread[myID] = d_array_current_Solution_thread[myID];
    solutions_block[myID]= myID;
    __syncthreads();
    while(salto){
        if(salto-(myID+1)>myID){
            if(solutions[myID]>solutions[salto-(myID+1)]){
                solutions[myID]=solutions[salto-(myID+1)];
                solutions_thread[myID]=solutions_thread[salto-(myID+1)];
                solutions_block[myID]=solutions_block[salto-(myID+1)];
            }
        }
        salto = (salto/2)+(salto&(2-1));
        if(salto==1){
            salto = 0;
        }
        __syncthreads();
    }
    if(myID==0)
    {
        
        d_array_current_Solution[myID] = solutions[myID];
        d_array_current_Solution_thread[myID]= solutions_thread[myID];
        d_array_current_Solution_block[myID] = solutions_block[myID];
    }
}



///////////////////////////////////////////////////
/// Parametros de configuración Default
///////////////////////////////////////////////////

double alpha1 = 15; // Alpha de distancia
double alpha2 = 30; // Alpha de segregación
double alpha3 = 25; // Alpha de costocupo
double coolingRate = 0.98; // Tasa de enfriamiento
double temp = 100000; // Temperatura inicial
double min_temp = 0.00000009; // Minima temperatura que puede llegar
int n_block = 256; // Numero de blockes = numeros de alumnos aleatorios
int n_thread = 85; // Numero de threads por bloque = numeros de escuelas aleatorios
std::string ruta_save = "./save/"; // Ruta para guardar los archivos
double k_recalentamiento = 0.90;
double max_temp = 0;
double e_const=0.01;
int count_rechaso=0;


///////////////////////////////////////////////////
/// Variables globales.
///////////////////////////////////////////////////
double alpha[3]={alpha1,alpha2,alpha3},
        **distMat=nullptr;
int n_students = 0,
        n_colegios,
        selectThread=0,
        selectBlock = 0,
        totalVuln = 0,
        *alumnosSep=nullptr,
        *cupoArray=nullptr;
int *previousSolution= nullptr;
int *bestSolution= nullptr;
int *currentSolution=nullptr;
int seed= 280;//rand();
double max_dist=0.0;
std::random_device rd;
std::mt19937 mt(rd());
std::uniform_int_distribution<int> dist(0,0);
std::uniform_int_distribution<int> dist2(0,0);


///////////////////////////////////////////////////
/// Funcion principal
///////////////////////////////////////////////////

int main(int argc, char *argv[]) {
    int  test_vectosize[29853];
    int  test_vectosize2[85*3];
    std::cout << "peso del vector" << sizeof(test_vectosize2) << "\n";

    time_t hora_actual;
    struct tm *time_info;
    time(&hora_actual);
    time_info = localtime(&hora_actual);
    char timestr[20];
    strftime(timestr, sizeof(timestr), "%Y-%m-%d T:%H-%M", time_info);
    std::string prefijo_save = std::string(timestr);

    if (argc>1) {
        alpha1 = std::stod(argv[1]); // Alpha de distancia
        alpha2 = std::stod(argv[2]); // Alpha de segregación
        alpha3 = std::stod(argv[3]); // Alpha de costocupo
        alpha[0]=alpha1;
        alpha[1]=alpha2;
        alpha[2]=alpha3;
        coolingRate = std::stod(argv[4]); // Tasa de enfriamiento
        k_recalentamiento = std::stod(argv[5]);
        temp = std::stod(argv[6]); // Temperatura inicial
        min_temp = std::stod(argv[7]); // Minima temperatura que puede llegar
        n_block = std::stoi(argv[8]); // Numero de blockes = numeros de alumnos aleatorios
        n_thread = std::stoi(argv[9]); // Numero de threads por bloque = numeros de escuelas aleatorios
        ruta_save = argv[10];
        prefijo_save = argv[11];
        max_temp= pow(10,300);
        seed= std::stoi(argv[12]);
    }
    mt.seed(seed);
    Info_colegio *ptr_colegios;
    Info_alu *ptr_students;




    std::ofstream info;
    std::string infotxt = ruta_save + prefijo_save + "-info.txt"; // concatenar
    info.open(infotxt);
    int x = 0, z = 0;

    ///////////////////////////////////////////////////
    /// Datos colegios
    /// Lee el archivo linea por linea y luego lo agrega al arreglo de estructura Info_colegio
    ///////////////////////////////////////////////////
    std::string line_colegios;
    std::ifstream info_school("colegios_utm.txt"); // concatenar
    std::getline(info_school, line_colegios);
    n_colegios = std::stoi(line_colegios);
    Info_colegio colegios[n_colegios];
    ptr_colegios = &colegios[0];
    while (std::getline(info_school, line_colegios)) {
        std::stringstream linestream(line_colegios);
        std::string data;
        std::getline(linestream, data, ',');
        ptr_colegios->rbd = std::stoi(data);
        std::getline(linestream, data, ',');
        ptr_colegios->latitude = std::stod(data);
        std::getline(linestream, data, ',');
        ptr_colegios->longitude = std::stod(data);
        std::getline(linestream, data, ',');
        ptr_colegios->num_alu = std::stoi(data);
        std::getline(linestream, data, ',');
        ptr_colegios->prioritario = std::stoi(data);
        ptr_colegios++;
    }

    ptr_colegios = &colegios[0]; // vuelve el puntero al inicio
    info_school.close();

    ///////////////////////////////////////////////////
    /// Datos Alumnos
    /// Lee el archivo linea por linea y luego lo agrega al arreglo de estructura info_student
    ///////////////////////////////////////////////////
    std::string line_student;
    std::ifstream info_student("alumnos_utm.txt"); // concatenar


    std::getline(info_student, line_student);
    n_students = std::stoi(line_student);
    Info_alu students[n_students];
    ptr_students = &students[0];
    while (std::getline(info_student, line_student)) {
        std::stringstream linestream(line_student);
        std::string data;
        std::getline(linestream, data, ',');
        ptr_students->rbd = std::stoi(data);
        std::getline(linestream, data, ',');
        ptr_students->latitude = std::stod(data);
        std::getline(linestream, data, ',');
        ptr_students->longitude = std::stod(data);
        std::getline(linestream, data, ',');
        ptr_students->sep = std::stoi(data);
        if (ptr_students->sep == 1) {
            totalVuln++;
        }
        ptr_students++;
    }
    ptr_students = &students[0]; // vuelve el puntero al inicio
    info_student.close();

    ///////////////////////////////////////////////////
    /// Se crea las soluciones que tienen n_students de largo
    ///////////////////////////////////////////////////
    previousSolution = (int *) malloc(sizeof(int) * n_students);
    bestSolution = (int *) malloc(sizeof(int) * n_students);



    ///////////////////////////////////////////////////
    /// Se asignan las escuelas un arreglo que y estudiantes a la escuela
    /// las escuelas tendran como identificación el indice
    /// y currentSolution tiene como indice al estudiante y el valor del indice a la escuela que asignada
    ///////////////////////////////////////////////////

    currentSolution = (int *) malloc(sizeof(int) * n_students);
    /// Se crea una matriz de distnacia donde se obtienen todas las distancias entre estudiantes y escuelas.
    distMat = (double **) malloc(sizeof(double) * n_students);
    cupoArray = (int *) malloc(sizeof(int) * n_colegios);
    ///Alumnos sep
    alumnosSep = (int *) malloc(sizeof(int) * n_students);
    /// Se crear un arreglo donde el el valor es la posición del estudiante sep
    for (x = 0; x < n_students; x++) {
        distMat[x] = (double *) malloc(sizeof(double) * n_colegios);
        alumnosSep[x] = students[x].sep;
    }

    assignSchoolToArray(ptr_colegios, ptr_students);
    calcDist(ptr_colegios, ptr_students);


    ///////////////////////////////////////////////////
    /// Termina La fase de recolección de datos.
    /// Es necesario crear una funcion que empareje al estudiante con la escuela correspondiente segun su puesto
    /// en el arreglo ejemplo el 5 estudiante tiene rbd 4566 ese apunta al colegio que esta en la posicion
    /// 20 entonces cambio el rbd del estudiante a 20
    ///////////////////////////////////////////////////

    double costBestSolution,
            costPreviousSolution,
            costCurrentSolution,
            sumaAlpha = 0;

    ///////////////////////////////////////////////////
    /// Calcula el valor de los alpha
    ///////////////////////////////////////////////////

    for (x = 0; x < 3; x++) {
        sumaAlpha += alpha[x];
    }

    for (x = 0; x < 3; x++) {
        alpha[x] = alpha[x] / (double) sumaAlpha;
    }

    ////////////////////////////////////////////////
    ////// Hace una calculo de rango de los promedios de las distancias
    ///////////////////////////////////////////////////

    for(int i=0;i<n_students;i++){
        for(x=0;x<n_colegios;x++){
            if(distMat[i][x]>max_dist){
                max_dist = distMat[i][x];
            }
        }
    }


    ///////////////////////////////////////////////////
    /// Registro de datos
    ///////////////////////////////////////////////////
    costBestSolution = calCosto(currentSolution);
    std::cout << "Primer costo de solución: " << costBestSolution << "\n";
    costPreviousSolution = costBestSolution;
    costCurrentSolution = costBestSolution;
    std::cout << "Primer distancia: " << meanDist(currentSolution) << "\n";


    std::cout << "Primer Segregación: " << S(currentSolution) << "\n";
    std::cout << "Primer CostoCupo: " << costCupo(currentSolution) << "\n";
    int count = 0;

    ///////////////////////////////////////////////////
    ///////////////////////////////////////////////////
    /// Reservando memoria en CUDA
    ///////////////////////////////////////////////////
    ///////////////////////////////////////////////////

    ///////////////////////////////////////////////////
    /// Matrices 2D
    ///////////////////////////////////////////////////
    double *d_distMat; /// clon de matriz de distancia
    int *d_currentSolution;
    int *d_alumnosSep; // Array que contendra a los estudiantes vulnerables

    ///////////////////////////////////////////////////
    /// Valores para las partes calcular la nueva solucion
    ///////////////////////////////////////////////////

    int aluVulxCol[n_colegios], aluxcol[n_colegios];
    int previousAluxCol[n_colegios];
    int previousAluVulxCol[n_colegios];
    int bestAluxCol[n_colegios];
    int bestAluVulxCol[n_colegios];

    for(x = 0; x < n_colegios; x++){
        aluxcol[x] = colegios[x].num_alu;
        previousAluxCol[x] = colegios[x].num_alu;
        bestAluxCol[x] = colegios[x].num_alu;
        aluVulxCol[x] = colegios[x].prioritario;
        previousAluVulxCol[x] = colegios[x].prioritario;
        bestAluVulxCol[x] = colegios[x].prioritario;

    }

    ///////////////
    double *d_array_current_Solution;
    int *d_array_current_Solution_thread;
    int *d_array_current_Solution_block;
    ///////////////
    int *d_aluxcol;
    int *d_aluVulxCol;
    int *d_shuffle_students;
    int *d_shuffle_colegios;




    hipMalloc((void **) &d_array_current_Solution, n_block * sizeof(double));
    hipMalloc((void **) &d_array_current_Solution_thread, n_block * sizeof(int));
    hipMalloc((void **) &d_array_current_Solution_block, sizeof(int));
    hipMalloc((void **) &d_shuffle_colegios, n_thread * sizeof(int));
    hipMalloc((void **) &d_shuffle_students, n_block * sizeof(int));
    hipMalloc((void **) &d_aluxcol,n_colegios * sizeof(int));
    hipMalloc((void **) &d_aluVulxCol,n_colegios * sizeof(int));
    hipMalloc((void **) &d_currentSolution, n_students * sizeof(int));  // Solución actual
    hipMalloc((void **) &d_alumnosSep, n_students * sizeof(int)); // arreglo que contiene la id de cada usuario vulnerable
    double *matrestest = (double *) malloc(sizeof(double) * n_students * n_colegios);
    double *array_costCurrentSolution = (double *) malloc(sizeof(double) * n_block * n_thread);
    for (x = 0; x < n_students; x++) {
        for (z = 0; z < n_colegios; z++) {
            matrestest[n_colegios * x + z] = distMat[x][z];
        }
    }
    for (x = 0; x < n_block; x++){
        for (z = 0; z < n_thread; z++){
            array_costCurrentSolution[n_thread * x + z] = 0.0;
        }
    }





    ///////////////////////////////////////////////////
    /// Valores que nunca van a cambiar
    //////////////////////////////////////////////////////

    hipMemcpy(d_alumnosSep, alumnosSep, n_students * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL( d_cupoArray), cupoArray,  n_colegios * sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL( d_alpha), alpha,  3 * sizeof(double));

    size_t pitch;
    hipMallocPitch(&d_distMat,
                    &pitch,
                    n_colegios * sizeof(double),
                    n_students); // Reserva memoria para la matriz de distancia

    size_t h_pitchBytes = n_colegios * sizeof(double);
    hipMemcpy2D(d_distMat,
                 pitch,
                 matrestest,
                 h_pitchBytes,
                 n_colegios * sizeof(double),
                 n_students,
                 hipMemcpyHostToDevice);


    ///////////////////////////////////////////////////
    /// Genera los archivos para graficos
    ///////////////////////////////////////////////////

    std::ofstream info_graficos;
    std::string name_info_graficos = ruta_save + prefijo_save +"-info-graficos.txt"; // concatenar
    info_graficos.open(name_info_graficos);
    info_graficos << count << "," << meanDist(currentSolution) << "," << S(currentSolution) << "," << costCupo(currentSolution) << "," << costCurrentSolution << "," << std::fixed <<  temp << std::setprecision(13) << "\n";
    std::cout << "Numero de bloques: " << n_block << "| Numeros de thread: "<< n_thread <<  "\n";
    strftime(timestr, sizeof(timestr), "%T", time_info);
    info << "Tiempo antes iniciar el ciclo" << prefijo_save << "\n";

    /// Posicion estudiantes

    std::ofstream info_graficos_bestSolution;
    std::string name_info_graficos_bestSolution = ruta_save + prefijo_save +"-info-graficos_bestSolution.txt"; // concatenar
    info_graficos_bestSolution.open(name_info_graficos_bestSolution);
    for(x=0;x<n_students;x++){
        info_graficos_bestSolution << currentSolution[x] << ",";
    }
    info_graficos_bestSolution << "\n";



    ///////////////////////////////////////////////////
    /// Genera arreglos que contendran valores del 0 hasta n_students y n_colegios
    ///////////////////////////////////////////////////
    int *shuffle_student = new int[n_students];
    int *shuffle_colegios = new int[n_colegios];
    for (int i = 0; i < n_students; i++) {
        shuffle_student[i] = i;
    }
    for (int i=0; i < n_colegios; i++){
        shuffle_colegios[i]=i;
    }

    ///////////////////////////////////////////////////
    /// Inicializa las distribuciónes
    ///////////////////////////////////////////////////

    dist = std::uniform_int_distribution<int>(0, n_students-1);
    dist2 = std::uniform_int_distribution<int>(0, n_colegios-1);

    ///////////////////////////////////////////////////
    /// Contador de tiempo de ejecución en cuda
    ///////////////////////////////////////////////////

    hipEvent_t start_cuda;
    hipEvent_t stop_cuda;
    hipEventCreate(&start_cuda);
    hipEventCreate(&stop_cuda);
    float elapsedTime;

    ///////////////////////////////////////////////////
    /// Inicio el contador de tiempo antes de iniciar el algortimo
    ///////////////////////////////////////////////////


    hipMemcpy(d_currentSolution, currentSolution, n_students * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_aluxcol, aluxcol, n_colegios * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_aluVulxCol, aluVulxCol, n_colegios * sizeof(int), hipMemcpyHostToDevice);





    std::vector<double> vector_costCurrentSolution;
    std::vector<double> vector_meanDist;
    std::vector<double> vector_segregation;
    std::vector<double> vector_costoCupo;
    std::vector<double> vector_temp;
    std::vector<int> vector_count;
    int reheating = 0;
    int c_accepta = 0;
    count++;
    double timeCuda = 0.0;

    int valmaxheating=25;
    int count_reheating = 0;
    double bestTemp = 0;

    double copytimecuda = 0;
    double executiontimecuda = 0;
    double recoverdatacuda = 0;
    ///////////////////////////////////////////////////
    /// Comienza a ejecutarse el algoritmo de SA
    ///////////////////////////////////////////////////
    auto start = std::chrono::high_resolution_clock::now();

    while(temp > min_temp){

        for(x=0;x<n_students;x++){
            currentSolution[x]=previousSolution[x];
        }
        for(x = 0; x < n_colegios; x++){
            aluxcol[x]=previousAluxCol[x];
            aluVulxCol[x]=previousAluVulxCol[x];
        }

        ///////////////////////////////////////////////////
        ///  Selecciona aleatoria mente a los alumnos
        ///////////////////////////////////////////////////

        shuffle(shuffle_student,n_block,dist);
        shuffle(shuffle_colegios,n_thread,dist2);

        ///////////////////////////////////////////////////
        /// Actualiza la memoria en CUDA
        ///////////////////////////////////////////////////


        hipMemcpy(d_currentSolution, currentSolution, n_students * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_aluxcol, aluxcol, n_colegios * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_aluVulxCol, aluVulxCol, n_colegios * sizeof(int), hipMemcpyHostToDevice);




        ///////////////////////////////////////////////////
        ///  Envia datos a GPU
        ///////////////////////////////////////////////////

        hipEventRecord(start_cuda,0);
        hipMemcpy(d_shuffle_students, shuffle_student, n_block * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_shuffle_colegios, shuffle_colegios, n_thread * sizeof(int), hipMemcpyHostToDevice);

        hipDeviceSynchronize();
        hipEventRecord(stop_cuda,0);
        hipEventSynchronize(stop_cuda);
        hipEventElapsedTime(&elapsedTime,start_cuda,stop_cuda);
        copytimecuda = copytimecuda+elapsedTime;

        ///////////////////////////////////////////////////
        ///  Ejecuta los kernel


        hipEventRecord(start_cuda,0);
        newSolution_kernel<<<n_block,n_thread,
                n_colegios* sizeof(int) + n_colegios* sizeof(int) + n_thread* sizeof(double)+ n_thread* sizeof(int)>>>(
                        d_array_current_Solution,
                                d_array_current_Solution_thread,
                                n_students,
                                n_colegios,
                                n_thread,
                                max_dist,
                                d_alumnosSep,
                                totalVuln,
                                d_aluxcol,
                                d_aluVulxCol,
                                d_currentSolution,
                                d_distMat,
                                d_shuffle_students,
                                d_shuffle_colegios,
                                pitch);
        hipDeviceSynchronize();

        reduce_block_kernel<<<1,n_block,
                n_block* sizeof(double)+ n_block* sizeof(int)+ n_block* sizeof(int)>>>(d_array_current_Solution,
                        d_array_current_Solution_thread,
                        d_array_current_Solution_block,
                        n_block);
        hipDeviceSynchronize();

        hipEventRecord(stop_cuda,0);
        hipEventSynchronize(stop_cuda);
        hipEventElapsedTime(&elapsedTime,start_cuda,stop_cuda);
        executiontimecuda = executiontimecuda+elapsedTime;

        hipDeviceSynchronize();
        hipEventRecord(start_cuda,0);
        hipMemcpy(&costCurrentSolution,&d_array_current_Solution[0], sizeof(double),hipMemcpyDeviceToHost);
        hipMemcpy(&selectThread,&d_array_current_Solution_thread[0], sizeof(int),hipMemcpyDeviceToHost);
        hipMemcpy(&selectBlock,d_array_current_Solution_block, sizeof(int),hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        hipEventRecord(stop_cuda,0);
        hipEventSynchronize(stop_cuda);
        hipEventElapsedTime(&elapsedTime,start_cuda,stop_cuda);
        recoverdatacuda = recoverdatacuda+elapsedTime;

        ///////////////////////////////////////////////////
        ///  Actualizo datos basicos
        ///////////////////////////////////////////////////

        //std::cout << "CPU: " << costCurrentSolution << " | " << selectThread << " | " << selectBlock << "\n";
        aluxcol[currentSolution[shuffle_student[selectBlock]]]-=1; ///
        aluVulxCol[currentSolution[shuffle_student[selectBlock]]]-=alumnosSep[shuffle_student[selectBlock]]; ///

        aluxcol[selectThread]+=1; ///
        aluVulxCol[selectThread]+=alumnosSep[shuffle_student[selectBlock]]; ///
        currentSolution[shuffle_student[selectBlock]] = selectThread; ///
        //aluxcol[shuffle_colegios[selectThread]]+=1; ///
        //aluVulxCol[shuffle_colegios[selectThread]]+=alumnosSep[shuffle_student[selectBlock]]; ///
        //currentSolution[shuffle_student[selectBlock]] = shuffle_colegios[selectThread]; ///




        ///////////////////////////////////////////////////
        /// Salida en caso de error
        ///////////////////////////////////////////////////
        //std::cout << costCurrentSolution << "\n";

        if(costCurrentSolution<0.00){
            std::cout << shuffle_colegios[selectThread] << "\n";
            std::cout << shuffle_student[selectBlock] << "\n";
            std::cout << "distancia: " << meanDist(currentSolution) << "\n";
            std::cout << "Segregación: " << S(currentSolution) << "\n";
            std::cout << "CostoCupo: " << costCupo(currentSolution) << "\n";
            std::cout << costCurrentSolution;
            exit(1);
        }






        if(costCurrentSolution < costBestSolution){
            // guarda la actual solución como la mejor
            for(x=0;x<n_students;x++){
                bestSolution[x]=currentSolution[x];
                previousSolution[x]=currentSolution[x];
            }
            for(x = 0; x < n_colegios; x++){
                previousAluxCol[x] = aluxcol[x];
                bestAluxCol[x] = aluxcol[x];
                previousAluVulxCol[x] = aluVulxCol[x];
                bestAluVulxCol[x] = aluVulxCol[x];

            }
            costBestSolution=costCurrentSolution;
            costPreviousSolution=costCurrentSolution;



            vector_costCurrentSolution.push_back(costCurrentSolution);
            vector_meanDist.push_back(meanDist(currentSolution));
            vector_segregation.push_back(S(currentSolution));
            vector_costoCupo.push_back(costCupo(currentSolution));
            vector_temp.push_back(temp);
            vector_count.push_back(count);
            c_accepta++;
            count_rechaso=0;
        }
        else{
            if(acepta(costPreviousSolution,costCurrentSolution)==1){
                /// Si por al asar acepta tomara la solución actual como la nueva solución a seguir
                for(x=0;x<n_students;x++){
                    previousSolution[x]=currentSolution[x];
                }
                for(x = 0; x < n_colegios; x++){
                    previousAluxCol[x] = aluxcol[x];
                    previousAluVulxCol[x] = aluVulxCol[x];
                }
                costPreviousSolution=costCurrentSolution;
                count_rechaso=0;
                c_accepta++;

            }
            else{
                count_rechaso++;
            }

        }
        ///////////////////////////////////////////////////
        /// Largo de temperatura
        ///////////////////////////////////////////////////
        if(c_accepta>=n_colegios){
            temp=temp*(coolingRate);
            //std::cout << "Enfriamiento " << temp << " CostZ " << costCurrentSolution  << " bestZ " <<  costBestSolution << " count_rechaso " << count_rechaso << " count_reheating " << count_reheating <<"\n";
            c_accepta=0;

        }
        if(count%((n_colegios*2))==0){
            temp=temp*(coolingRate);
            //std::cout << "recalentamiento " << temp << " CostZ " << costCurrentSolution << " bestZ " << costBestSolution << " count_rechaso " << count_rechaso << " count_reheating " << count_reheating <<"\n";
        }

        ///////////////////////////////////////////////////
        /// Reinicio de temperatura
        ///////////////////////////////////////////////////

        count++;
    }

    ///////////////////////////////////////////////////
    /// Obtiene el tiempo de ejecución
    ///////////////////////////////////////////////////
    auto end = std::chrono::high_resolution_clock::now();
    double time_taken = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
    time_taken *= 1e-9;

    for(x=0;x<n_students;x++){
        info_graficos_bestSolution << bestSolution[x] << ",";
    }
    info_graficos_bestSolution.close();


    for(x=0; x<vector_count.size(); x++){
        info_graficos << vector_count.at(x) << "," << vector_meanDist.at(x) << "," << vector_segregation.at(x) << "," << vector_costoCupo.at(x) << "," << vector_costCurrentSolution.at(x) << "," << std::fixed << vector_temp.at(x) << std::setprecision(13) << ","<< max_dist << "\n";
    }
    info_graficos.close();
    info << "Tiempo despues de finalizar el ciclo" << prefijo_save << "\n";
    std::cout << "--------------- Resultado Final ----------------" << "\n";
    info  << "--------------- Resultado Final con restauración----------------" << "\n";
    std::cout << "Numero de Ciclos: " << count << "\n";
    info << "Numero de Ciclos: " << count << "\n";
    std::cout << "Costo de la solución previa: " << costPreviousSolution << "\n";
    info << "Costo de la solución previa: " << costPreviousSolution << "\n";
    std::cout << "Costo de la mejor solución: " << costBestSolution << "\n";
    info << "Costo de la mejor solución: " << costBestSolution << "\n";
    std::cout << "Costo de la solución actual: " << costCurrentSolution << "\n";
    info << "Costo de la solución actual: " << costCurrentSolution << "\n";
    std::cout << "Tiempo de ejecución de SA: " << std::fixed << time_taken << std::setprecision(9) << "\n";
    info << "Tiempo de ejecución de SA: " << std::fixed << time_taken << std::setprecision(9) << "\n";
    std::cout << "distancia: " << meanDist(bestSolution) << "\n";
    info << "distancia: " << meanDist(bestSolution) << "\n";
    std::cout << "Segregación: " << S(bestSolution) << "\n";
    info << "Segregación: " << S(bestSolution) << "\n";
    std::cout << "CostoCupo: " << costCupo(bestSolution) << "\n";
    info << "CostoCupo: " << costCupo(bestSolution) << "\n";
    std::cout << "distancia * alpha: " << meanDist(bestSolution) * alpha[0] << "\n";
    info << "distancia * alpha: " << meanDist(bestSolution) * alpha[0] << "\n";
    std::cout << "Segregación * alpha: " << S(bestSolution) * alpha[1] << "\n";
    info << "Segregación * alpha: " << S(bestSolution) * alpha[1] << "\n";
    std::cout << "CostoCupo * alpha: " << costCupo(bestSolution) * alpha[2] << "\n";
    info << "CostoCupo * alpha: " << costCupo(bestSolution) * alpha[2] << "\n";
    std::cout << "solución calculada aparte: " << calCosto(bestSolution) << "\n";
    info << "solución calculada aparte: " << calCosto(bestSolution) << "\n";
    info << "Numero de bloques: " << n_block << "| Numeros de thread: "<< n_thread <<  "\n";
    std::cout << "Numero de bloques: " << n_block << "| Numeros de thread: "<< n_thread <<  "\n";
    std::cout << "--------------- Finalizo con exito ----------------" << "\n";
    info << "--------------- Finalizo con exito ----------------" << "\n";
    info.close();

    std::cout << "-------------- Guardando Archivos /cmake-build-dbug-save -----------------" << "\n";



    std::ofstream studentscsv,schoolcsv, info_test;

    std::string nameinfo_test = ruta_save + prefijo_save +"-info_test.txt"; // concatenar
    info_test.open(nameinfo_test);
    info_test << std::fixed << time_taken << std::setprecision(9) << "," << costBestSolution << "," << meanDist(bestSolution) << "," << S(bestSolution) << "," << costCupo(bestSolution) << ","<< n_block << "," << n_thread << "," << count << "," << std::fixed << temp << std::setprecision(13) << "," << min_temp << "," << coolingRate << "," << k_recalentamiento << "," << alpha1 << "," << alpha2 << "," << alpha3 << "," << seed <<  "," << timeCuda/1000 << "\n";
    info_test.close();

    std::cout << "Tiempo de copia: "<< copytimecuda << "\n";
    std::cout << "Tiempo de proceso: " <<  executiontimecuda << "\n";
    std::cout << "Tiempo de rescate de datos: " << recoverdatacuda << "\n";
    std::cout << "-------------- Archivos Guardado ------------------" << "\n";

    hipFree(d_currentSolution);
    hipFree(d_alumnosSep);
    hipFree(d_distMat);
    hipFree(d_cupoArray);
    hipFree(d_array_current_Solution);
    hipFree(d_array_current_Solution_thread);
    hipFree(d_array_current_Solution_block);
    hipFree(d_alpha);
    hipEventDestroy(start_cuda);
    hipEventDestroy(stop_cuda);
    return (EXIT_SUCCESS);

}

///////////////////////////////////////////////////
/// Calcula el costo
///////////////////////////////////////////////////
double calCosto(const int currentThreadSolution[]){
    double var1 = meanDist(currentThreadSolution)/max_dist;
    std::cout << "distancia: " << var1 << "\n";
    double var2 = S(currentThreadSolution);
    std::cout << "Segregación: " << var2 << "\n";
    double var3 = costCupo(currentThreadSolution);
    std::cout << "CostoCupo: " << var3 << "\n";
    return (double)((alpha[0]*var1)+(alpha[1]*var2)+(alpha[2]*var3));
}

///////////////////////////////////////////////////
/// Distancia promedio que recorreran los estudiantes
///////////////////////////////////////////////////
double meanDist(const int currentThreadSolution[]){
    double sumDist=0;
    for(int i=0;i<n_students;i++){
        sumDist+=distMat[i][currentThreadSolution[i]]; // distMat[estudiante][escuela]
    }
    double mean=sumDist/double(n_students);

    //std::cout << "Numero de estudiantes: " << n_student << "  |  Suma de distancias:" << sumDist << "\n";
    return mean;
}
///////////////////////////////////////////////////
/// Calcula segregación por duncan
///////////////////////////////////////////////////
double S(const int currentThreadSolution[]){
    double totalSesc = 0.0;
    int aluVulCol =0;
    int aluNoVulCol = 0;
    for(int n=0; n<n_colegios;n++) {
        aluVulCol = 0;
        aluNoVulCol = 0;
        for (int a = 0; a < n_students; a++) {
            if (currentThreadSolution[a] == n) {
                aluNoVulCol++;
                aluVulCol+=alumnosSep[a];
            }
        }
        if(aluNoVulCol>0){
            aluNoVulCol =aluNoVulCol - aluVulCol;
            totalSesc+=((double)1/2)*fabs((aluVulCol/(double)totalVuln)-(aluNoVulCol/(double)(n_students-totalVuln)));
        }
    }
    return totalSesc;
}
///////////////////////////////////////////////////
/// Calcula el costo de tener los estudiantes en las escuelas
///////////////////////////////////////////////////
double costCupo(const int currentThreadSolution[]){
    double totalcostCupo = 0;
    int totalAluCol = 0;
    for(int j=0;j<n_colegios;j++){
        totalAluCol = 0;
        for(int i=0; i<n_students; i++){
            if(currentThreadSolution[i]==j){
                totalAluCol++;
            }
        }
        totalcostCupo+=totalAluCol*fabs((cupoArray[j]-totalAluCol)/pow(((double)cupoArray[j]/2),2));
    }
    return (totalcostCupo/n_colegios);
}
///////////////////////////////////////////////////
/// Función de aceptación en base a mayor temperatura mayor probabilidad que acepte a una solución peor
/// en caso de menor temperatura menor probabibilidad que acepte una solución peor.
///////////////////////////////////////////////////
int acepta(double costPrevious, double costCurrent){;
    std::uniform_real_distribution<double> dist_accepta(0.0, 1.0);
    if(costCurrent < costPrevious){
        return 1;
    }
    else{
        double valor=p(costPrevious,costCurrent);
        double nrandom=dist_accepta(mt);
        if(nrandom<valor){
            return 1;
        }
        else{
            return 0;
        }
    }
}
double p(double costPrevious,double costCurrent){
    double po;
    po = exp(-(costCurrent-costPrevious)/((double)temp));
    //po = 1/(1+exp(-(costPrevious-costCurrent)/temp));
    return po;
}

///////////////////////////////////////////////////
/// Asigna a las soluciones la escuela actual Solo se utiliza al inicio
///////////////////////////////////////////////////
void assignSchoolToArray(Info_colegio *ptr_colegios, Info_alu *ptr_students){
    Info_alu *ptr_aux = ptr_students;
    for(int x=0;x < n_colegios;x++){
        for(int y=0; y < n_students; y++){
            if(ptr_colegios->rbd == ptr_students->rbd){
                previousSolution[y] = x;
                bestSolution[y] = x;
                currentSolution[y] = x;
            }
            ptr_students++;

        }
        /*
         * cupoArray sera un arreglo que por indice es la escuela y su valor sera el cupo que posee esa escuela
         * se asume que las escuelas pueden tener sobre cupo.
         */
        //std::cout << ptr_colegios->num_alu+3 << "\n";
        cupoArray[x] = ptr_colegios->num_alu+ ((int)((ptr_colegios->num_alu*10)/100));
        ptr_students = ptr_aux;
        ptr_colegios++;
    }
}

///////////////////////////////////////////////////
/// Crea una matriz de distancia donde x es el estudiante, y es la escuela
///////////////////////////////////////////////////
void calcDist(Info_colegio *ptr_colegios, Info_alu *ptr_students){
    Info_colegio *ptr_aux = ptr_colegios;
    for(int x=0;x < n_students ;x++){
        for(int y=0; y < n_colegios; y++){
            distMat[x][y] = sqrt( pow((ptr_students->latitude - ptr_colegios->latitude),2)+pow((ptr_students->longitude - ptr_colegios->longitude),2))/1000;
            ptr_colegios++;
        }
        ptr_colegios = ptr_aux;
        ptr_students++;
    }
}

void shuffle(int values[], const int max_change, std::uniform_int_distribution<int> distri) {
    int randvalue1,randvalue2,tem_value;
    for (int i = 0; i<max_change; i++) {
        randvalue1 = distri(mt);
        randvalue2 = i;
        tem_value = values[randvalue1];
        values[randvalue1] = values[randvalue2];
        values[randvalue2] = tem_value;
    }
    
}


///

//////// convierte la distancia
//// exp(x*fabs(((x-init)/max_dist)-init)/max_dist-min_dist)*exp(-(max_dist*fabs(((max_dist-init)/max_dist)-init)/max_dist-min_dist))